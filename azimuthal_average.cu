#include "hip/hip_runtime.h"
//////////////////////////////////////
//  Reduction code is based heavily on the reduction_example from Nvidia's CUDA SDK examples
//  See "Optimizing parallel reduction in CUDA" - M. Harris for more details
//  some tweaks in regard to adding Boolean mask made
//////////////////////////////////////

#include <string>
#include <iostream>
#include <fstream>
#include <algorithm> 
#include <vector>

#include "constants.hpp"
#include "debug.hpp"
#include "azimuthal_average_kernel.cuh"

inline unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

///////////////////////////////////////////////////////
//	Writes ISF(lambda, tau) to file. 
//  When angle analysis is enabled, it writes separate
//  ISF data for each angular segment, including angle
//  information in the output file.
///////////////////////////////////////////////////////
void writeIqtToFile(std::string filename,
                    float *ISF,
                    float *lambda_arr, int lambda_count,
                    int   *tau_arr,    int tau_count,
                    float fps,
                    bool enable_angle_analysis,
                    int angle_count) {

    std::ofstream out_file(filename); 

    if (out_file.is_open()) {
        // lambda - values
        for (int lidx = 0; lidx < lambda_count; lidx++) {
            out_file << lambda_arr[lidx] << " ";
        }
        out_file << "\n";

        // tau - values
        for (int ti = 0; ti < tau_count; ti++) {
            out_file << static_cast<float>(tau_arr[ti]) / fps << " ";
        }
        out_file << "\n";

        if (enable_angle_analysis) {
            int full_angle_count = 2 * angle_count;  
            for (int angle_idx = 0; angle_idx < full_angle_count; angle_idx++) {
                
                float angle_width = 180.0 / angle_count;
                out_file << "# Angle section " + std::to_string(angle_idx) + 
                    " (center angle " + std::to_string((angle_idx * angle_width - 90.0) + angle_width/2) + 
                    " degrees, range: " + std::to_string(angle_idx * angle_width - 90.0) + "-" + 
                    std::to_string((angle_idx + 1) * angle_width - 90.0) + " degrees)\n";

                // I(lambda, tau) - values
                for (int li = 0; li < lambda_count; li++) {
                    for (int ti = 0; ti < tau_count; ti++) {
                        int idx = (angle_idx < angle_count) 
                                  ? (li * angle_count + angle_idx) * tau_count + ti
                                  : (li * angle_count + (angle_idx + full_angle_count / 2) % full_angle_count) * tau_count + ti;
                        out_file << ISF[idx] << " ";
                    }
                    out_file << "\n";
                }
                out_file << "\n";
            }
        } else {
            for (int li = 0; li < lambda_count; li++) {
                for (int ti = 0; ti < tau_count; ti++) {
                    out_file << ISF[li * tau_count + ti] << " ";
                }
                out_file << "\n";
            }
        }

        out_file.close();
        verbose("I(lambda, tau) written to %s\n", filename.c_str());
    } else {
        fprintf(stderr, "[Out Error] Unable to open %s.\n", filename.c_str());
        exit(EXIT_FAILURE);
    }
}


// Device analysis

///////////////////////////////////////////////////////
//	This function builds azimuthal boolean pixel masks
//	based on given input parameters. When angle analysis
//  is enabled, it creates separate masks for different
//  angular segments at each q-value. Masks are built on
//	host and copied to device memory location.
///////////////////////////////////////////////////////
void buildAzimuthMask(bool *d_mask_out,
                      int *h_pixel_counts,
                      float *q_arr, int q_count,
                      float q_tolerance,
                      int w, int h,
                      bool enable_angle_analysis,
                      int angle_count) {
    
    int total_masks = q_count * (enable_angle_analysis ? angle_count : 1);
    
    float q2_arr[q_count]; // array containing squared q-values
    for (int i = 0; i < q_count; i++)
        q2_arr[i] = q_arr[i] * q_arr[i];
    
    int element_count = (w/2 + 1) * h; // number of elements in mask (right half of FFT)
    bool *h_mask = new bool[element_count * total_masks];
    memset(h_mask, 0, sizeof(bool) * element_count * total_masks);
    
    // pre-calc some values
    float tol2 = q_tolerance * q_tolerance;
    int half_w = w / 2;
    int half_h = h / 2;
    
    int x_shift, y_shift;
    float r2, r2q2_ratio;

    bool px;
    float angle_step = M_PI / angle_count;  // Size of each angular segment
    
    // Initialize pixel counts for all masks to zero
    for (int mask_idx = 0; mask_idx < total_masks; mask_idx++) {
        h_pixel_counts[mask_idx] = 0;
    }
    
    // Create masks for each q-value
    for (int q_idx = 0; q_idx < q_count; q_idx++) {
        // Iterate over each pixel in the right half of the image
        for (int x = 0; x < (w/2 + 1); x++) {
            for (int y = 0; y < h; y++) {
                // Calculate pixel offset from center (FFT shift)
                x_shift = (x + half_w) % w - half_w;
                y_shift = (y + half_h) % h - half_h;
                
                r2 = x_shift * x_shift + y_shift * y_shift;
                r2q2_ratio = r2 / q2_arr[q_idx];
                
                // Check if pixel is within the annular region for this q-value
                px = (1 <= r2q2_ratio) && (r2q2_ratio <= tol2);
                
                if (px) {  // If pixel is within the annular region
                    if (enable_angle_analysis) {
                        // Calculate pixel angle (-π to π range)
                        float angle = atan2(y_shift, x_shift);
                        
                        // Normalize angle to 0-1 range (mapping -π/2 to 0, π/2 to 1)
                        float normalized_angle = (angle + M_PI/2) / M_PI;
                        
                        // Determine which angular segment the pixel belongs to
                        int angle_idx = std::min((int)(normalized_angle * angle_count), angle_count - 1);
                        
                        // Calculate mask index for this q-value and angle segment
                        int mask_idx = q_idx * angle_count + angle_idx;
                        
                        // Update corresponding mask and count
                        h_mask[mask_idx * element_count + y * (w/2 + 1) + x] = true;
                        h_pixel_counts[mask_idx]++;
                    } else {
                        // No angle analysis - just count pixels for this q-value
                        h_pixel_counts[q_idx]++;
                        h_mask[q_idx * element_count + y * (w/2 + 1) + x] = px;
                    }
                } else if (!enable_angle_analysis) {
                    // When angle analysis is disabled and pixel is not in range, set mask to false
                    h_mask[q_idx * element_count + y * (w/2 + 1) + x] = false;
                }
            }
        }
    }

    // Check if each mask has pixels meeting the criteria
    if (enable_angle_analysis) {
        for (int q_idx = 0; q_idx < q_count; q_idx++) {
            for (int angle_idx = 0; angle_idx < angle_count; angle_idx++) {
                int mask_idx = q_idx * angle_count + angle_idx;
                if (h_pixel_counts[mask_idx] == 0) {
                    verbose("[Mask Generation] q: %f, (#q: %d, angle: %d) has zero mask pixels for scale %d x %d\n", 
                           q_arr[q_idx], q_idx, angle_idx, w, h);
                }
            }
        }
    } else {
        // No angle analysis - check masks for each q-value only
        for (int q_idx = 0; q_idx < q_count; q_idx++) {
            if (h_pixel_counts[q_idx] == 0) {
                verbose("[Mask Generation] q: %f, (#q: %d) has zero mask pixels for scale %d x %d\n", q_arr[q_idx], q_idx, w, h);
            }
        }
    }

    // Copy mask onto GPU
    gpuErrorCheck(hipMemcpy(d_mask_out, h_mask, sizeof(bool) * element_count * total_masks, hipMemcpyHostToDevice));

    delete[] h_mask;  
}

///////////////////////////////////////////////////////
// Code to perform masked (GPU) reduction of ISF
// Analyzes FFT data using boolean masks to compute the ISF
// When angle analysis is enabled, processes separate masks
// for different angular segments at each q-value
// For optimal performance with reduction operations,
// future work could consider performing two separate reductions
// on (w/2)*(h/2) blocks which are more likely to be powers of 2
///////////////////////////////////////////////////////
float * analyseFFTDevice(float *d_data_in,
                        bool *d_mask,
                        int *h_px_count,
                        float norm_factor,
                        int tau_count,
                        int q_count,
                        int tile_count,
                        int w, int h,
                        bool enable_angle_analysis,
                        int angle_count) {

    // Total elements in the right half of the FFT
    int n = (w / 2 + 1) * h;

    // Compute the number of threads and blocks for the reduction kernel
    // For small datasets, use power of 2 sized threadblocks scaled to data size
    // For larger datasets, use fixed BLOCKSIZE (defined in header)
    int threads = (n < BLOCKSIZE * 2) ? nextPow2((n + 1) / 2) : BLOCKSIZE;
    int blocks = (n + (threads * 2 - 1)) / (threads * 2);

    // Limit maximum blocks to 64 for optimal GPU scheduling
    // This is a performance tuning parameter that may vary by GPU architecture
    blocks = (64 < blocks) ? 64 : blocks;

    // Allocate device and host memory for intermediate reduction results
    float *d_intermediateSums;
    float *h_intermediateSums = new float[blocks];

    gpuErrorCheck(hipMalloc((void **)&d_intermediateSums, sizeof(float) * blocks));

    // Calculate total number of q-values to process
    // For angle analysis, this is q_count * angle_count (covering half-circle)
    int total_q = q_count;
    if (enable_angle_analysis) {
        total_q = q_count * angle_count; // angle_count represents half-circle segments
    }
    
    // Allocate and initialize ISF result array (tau_count values for each q-angle combination)
    float * ISF = new float[tau_count * total_q]();
    
    // Process each tau value
    for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
        if (enable_angle_analysis) {
            // With angle analysis: process each q-value and angle segment combination
            for (int q_idx = 0; q_idx < q_count; q_idx++) {
                for (int angle_idx = 0; angle_idx < angle_count; angle_idx++) {
                    int mask_idx = q_idx * angle_count + angle_idx;
                    float val = 0;
                    
                    // Only process if the mask contains pixels
                    if (h_px_count[mask_idx] != 0) {
                        // Execute the reduction kernel for this mask and tau value
                        maskReduce<float>(n, threads, blocks, d_data_in + n*tau_idx*tile_count, 
                                         d_mask + n*mask_idx, d_intermediateSums);
                        
                        // Copy partial results from device to host
                        gpuErrorCheck(hipMemcpy(h_intermediateSums, d_intermediateSums, blocks * sizeof(float), hipMemcpyDeviceToHost));

                        // Combine partial sums from each block on CPU
                        for (int i = 0; i < blocks; i++) {
                            val += h_intermediateSums[i];
                        }

                        // Normalize by pixel count and apply normalization factor
                        val /= static_cast<float>(h_px_count[mask_idx]);
                        val *= norm_factor;
                    }
                    
                    // Store result in ISF array
                    ISF[mask_idx * tau_count + tau_idx] = val;
                }
            }
        } else {
            // Without angle analysis: process each q-value only
            for (int q_idx = 0; q_idx < q_count; q_idx++) {
                float val = 0;
                
                // Only process if the mask contains pixels
                if (h_px_count[q_idx] != 0) {
                    // Execute the reduction kernel for this mask and tau value
                    maskReduce<float>(n, threads, blocks, d_data_in + n*tau_idx*tile_count, 
                                     d_mask + n*q_idx, d_intermediateSums);
                    
                    // Copy partial results from device to host
                    gpuErrorCheck(hipMemcpy(h_intermediateSums, d_intermediateSums, blocks * sizeof(float), hipMemcpyDeviceToHost));

                    // Combine partial sums from each block on CPU
                    for (int i = 0; i < blocks; i++) {
                        val += h_intermediateSums[i];
                    }
                    
                    // Multiply by 2 to account for FFT symmetry (only processing right half)
                    val *= 2;
                    // Normalize by pixel count and apply normalization factor
                    val /= static_cast<float>(h_px_count[q_idx]);
                    val *= norm_factor;
                }
                
                // Store result in ISF array
                ISF[q_idx * tau_count + tau_idx] = val;
            }
        }
    }

    // Free temporary device and host memory to prevent memory leaks
    hipFree(d_intermediateSums);
    delete[] h_intermediateSums;

    // Ensure all GPU operations are complete before returning
    hipDeviceSynchronize();

    // Return the computed ISF array (caller is responsible for freeing this memory)
    return ISF;
}
